#include <stdio.h>
#include "all_structure.cuh"
#include "gpuFunctions.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include<vector>
#include <chrono> 


#define THREADS_PER_BLOCK 1024 //we can change it

using namespace std;
using namespace std::chrono;


/*
1st arg: original graph file name
2nd arg: no. of nodes     
3rd arg: no. of edges    
4th arg: input SSSP file name
5th arg: change edges file name
****main commands to run****
nvcc -o op main2.cu
./op <fullgraph file name> <SSSP file name> <changeEdges file name> <no. of nodes> <no. of edges * 2 (or total number of lines in fullgraph file)>
*/
int main(int argc, char* argv[]) {

	int nodes, edges;
	hipError_t cudaStatus;
	nodes = atoi(argv[2]);
	edges = atoi(argv[3]);
	int deviceId;
	int numberOfSMs;

	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
	
	//Reading Original input graph
	vector<ColWtList> InEdgesList;
	InEdgesList.resize(nodes);
	int* InEdgesListTracker = (int*)malloc((nodes + 1) * sizeof(int));//we take nodes +1 to store the start ptr of the first row 
	vector<ColWt> InEdgesListFull;
	vector<ColWtList> OutEdgesList;
	OutEdgesList.resize(nodes);
	int* OutEdgesListTracker = (int*)malloc((nodes + 1) * sizeof(int));//we take nodes +1 to store the start ptr of the first row 
	vector<ColWt> OutEdgesListFull;
	read_graphEdges(InEdgesList, argv[1], &nodes,  OutEdgesList);
	//Reading change edges input
	vector<changeEdge> allChange;
	readin_changes(argv[5], allChange, InEdgesList, OutEdgesList);

	//create 1D array from 2D to fit it in GPU
	InEdgesListTracker[0] = 0; //start pointer points to the first index of InEdgesList
	OutEdgesListTracker[0] = 0; //start pointer points to the first index of OutEdgesList
	for (int i = 0; i < nodes; i++)
	{
		InEdgesListTracker[i + 1] = InEdgesListTracker[i] + InEdgesList.at(i).size();
		InEdgesListFull.insert(std::end(InEdgesListFull), std::begin(InEdgesList.at(i)), std::end(InEdgesList.at(i)));
		OutEdgesListTracker[i + 1] = OutEdgesListTracker[i] + OutEdgesList.at(i).size();
		OutEdgesListFull.insert(std::end(OutEdgesListFull), std::begin(OutEdgesList.at(i)), std::end(OutEdgesList.at(i)));
	}



	//Transferring input graph and change edges data to GPU
	ColWt* InEdgesListFull_device;
	cudaStatus = hipMallocManaged(&InEdgesListFull_device, edges * sizeof(ColWt));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at InEdgesListFull structure");
	}
	std::copy(InEdgesListFull.begin(), InEdgesListFull.end(), InEdgesListFull_device);

	ColWt* OutEdgesListFull_device;
	cudaStatus = hipMallocManaged(&OutEdgesListFull_device, edges * sizeof(ColWt));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at InEdgesListFull structure");
	}
	std::copy(OutEdgesListFull.begin(), OutEdgesListFull.end(), OutEdgesListFull_device);
	
	int* InEdgesListTracker_device;
	cudaStatus = hipMalloc((void**)&InEdgesListTracker_device, nodes * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at InEdgesListTracker_device");
	}
	hipMemcpy(InEdgesListTracker_device, InEdgesListTracker, nodes * sizeof(int), hipMemcpyHostToDevice);
	int* OutEdgesListTracker_device;
	cudaStatus = hipMalloc((void**)&OutEdgesListTracker_device, nodes * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at OutEdgesListTracker_device");
	}
	hipMemcpy(OutEdgesListTracker_device, OutEdgesListTracker, nodes * sizeof(int), hipMemcpyHostToDevice);
	
	int totalChangeEdges = allChange.size();
	changeEdge* allChange_device;
	cudaStatus = hipMallocManaged(&allChange_device, totalChangeEdges * sizeof(changeEdge));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at allChange structure");
	}
	std::copy(allChange.begin(), allChange.end(), allChange_device);
	//set hipMemAdviseSetReadMostly by the GPU for change edge data
	hipMemAdvise(allChange_device, totalChangeEdges * sizeof(changeEdge), hipMemAdviseSetReadMostly, deviceId);


	//Test start
	/*cout << "change edges:" << endl;
	for (int i = 0; i < totalChangeEdges; i++)
	{
		cout << allChange_device[i].node1 << " " << allChange_device[i].node2 << " " << allChange_device[i].edge_wt << " " << allChange_device[i].inst << endl;
	}
	cout << "In edges in Unified memory" << endl;
	for (int i = 0; i < nodes; i++)
	{
		cout << "row: " << i << endl;
		for (int j = InEdgesListTracker[i]; j < InEdgesListTracker[i + 1]; j++)
		{
			cout << InEdgesListFull_device[j].col << " : " << InEdgesListFull_device[j].wt << endl;
		}
	}
	cout << "Out edges in Unified memory" << endl;
	for (int i = 0; i < nodes; i++)
	{
		cout << "row: " << i << endl;
		for (int j = OutEdgesListTracker[i]; j < OutEdgesListTracker[i + 1]; j++)
		{
			cout << OutEdgesListFull_device[j].col << " : " << OutEdgesListFull_device[j].wt << endl;
		}
	}*/
	//Test end
	
	//Reading SSSP Tree input and storing directly on unified memory
	RT_Vertex* SSSP;
	cudaStatus = hipMallocManaged(&SSSP, nodes * sizeof(RT_Vertex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSP structure");
	}
	read_SSSP(SSSP, argv[4], &nodes);
	//set hipMemAdviseSetPreferredLocation at GPU for SSSP data
	hipMemAdvise(SSSP, nodes * sizeof(RT_Vertex), hipMemAdviseSetPreferredLocation, deviceId);



	//test start
	/*cout << "after reading SSSP:" << endl;
	for (int i = 0; i < nodes; i++)
	{
		cout << "row: " << i << " dist: " << SSSP[i].Dist << " parent: " << SSSP[i].Parent << endl;

	}*/
	//test end



	//double inf = std::numeric_limits<double>::infinity();
	int inf = 999999;
	int* Edgedone;
	hipMallocManaged(&Edgedone, (totalChangeEdges) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSP structure");
	}
	//set hipMemAdviseSetPreferredLocation at GPU for SSSP data
	hipMemAdvise(Edgedone, (totalChangeEdges) * sizeof(int), hipMemAdviseSetPreferredLocation, deviceId);
	//Asynchronous prefetching of data
	hipMemPrefetchAsync(Edgedone, (totalChangeEdges) * sizeof(int), deviceId);
	hipMemPrefetchAsync(allChange_device, totalChangeEdges * sizeof(changeEdge), deviceId);
	hipMemPrefetchAsync(SSSP, nodes * sizeof(RT_Vertex), deviceId);
	hipMemPrefetchAsync(InEdgesListFull_device, edges * sizeof(ColWt), deviceId);
	hipMemPrefetchAsync(OutEdgesListFull_device, edges * sizeof(ColWt), deviceId);
	//initialize Edgedone array with -1
	initializeEdgedone << <(totalChangeEdges / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (Edgedone, totalChangeEdges);
	deleteEdge << < (totalChangeEdges / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (allChange_device, Edgedone, SSSP, totalChangeEdges, inf, InEdgesListFull_device, OutEdgesListFull_device, InEdgesListTracker_device, OutEdgesListTracker_device);
	insertEdge << < (totalChangeEdges / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (allChange_device, Edgedone, SSSP, totalChangeEdges, inf, InEdgesListFull_device, OutEdgesListFull_device, InEdgesListTracker_device, OutEdgesListTracker_device);
	
	
	
	//test start
	/*hipDeviceSynchronize();
	cout << "\nafter insertDeleteEdge SSSP:" << endl;
	
	for (int i = 0; i < nodes; i++)
	{
		cout << "row: " << i << " dist: " << SSSP[i].Dist << " parent: " << SSSP[i].Parent << endl;

	}*/
	//test end





	//Go over the inserted edges to see if they need to be changed. Correct edges are connected in this stage
	int* change_d = new int[1];
	int* change = new int[1];
	change[0] = 1;
	hipMalloc((void**)&change_d, 1 * sizeof(int));
	while (change[0] == 1)
	{
		change[0] = 0;
		hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);
		checkInsertedEdges << < (totalChangeEdges / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (allChange_device, totalChangeEdges, Edgedone, SSSP, change_d);
		hipMemcpy(change, change_d, 1 * sizeof(int), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();

	}

	hipFree(Edgedone); //free memory before neighbor update

	int its = 0;
	change[0] = 1;
	while (change[0] == 1 && its < 70)
	{
		//printf("Iteration:%d \n", its);

		change[0] = 0;
		hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);
		updateNeighbors << <(nodes / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (SSSP, nodes, inf, InEdgesListFull_device, OutEdgesListFull_device, InEdgesListTracker_device, OutEdgesListTracker_device, change_d);
		hipMemcpy(change, change_d, 1 * sizeof(int), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		its++;
	}//end of while
	printf("Total Iterations to Converge %d \n", its);

	for (int i = 0; i < nodes; i++)
	{
		cout << "row: " << i << " dist: " << SSSP[i].Dist <<" parent: " << SSSP[i].Parent << endl;
		
	}



	hipFree(change_d);
	hipFree(InEdgesListTracker_device);
	hipFree(OutEdgesListTracker_device);
	hipFree(OutEdgesListFull_device);
	hipFree(InEdgesListFull_device);
	hipFree(allChange_device);
	hipFree(SSSP);
	return 0;
}


