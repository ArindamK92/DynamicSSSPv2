#include <stdio.h>
#include "all_structure_undir.cuh"
#include "gpuFunctions_undir.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include<vector>
#include <chrono>
#include <algorithm>

#define THREADS_PER_BLOCK 1024 //we can change it

using namespace std;
using namespace std::chrono;


/*
1st arg: original graph file name
2nd arg: no. of nodes
3rd arg: no. of edges
4th arg: input SSSP file name
5th arg: change edges file name
****main commands to run****
nvcc -o op_main CudaSSSPmain.cu
./op_main original_graph_file_name number_of_nodes number_of_edges input_SSSP_file_name change_edge_file_name
*/
int main(int argc, char* argv[]) {

	int nodes, edges;
	hipError_t cudaStatus;
	nodes = atoi(argv[2]);
	edges = atoi(argv[3]);
	int deviceId;
	int numberOfSMs;
	int totalInsertion = 0;

	//Get gpu device id and number of SMs
	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	//Reading Original input graph
	vector<ColWtList> AdjList;
	AdjList.resize(nodes);
	int* AdjListTracker = (int*)malloc((nodes + 1) * sizeof(int));//we take nodes +1 to store the start ptr of the first row
	vector<ColWt> AdjListFull;
	cout << "Reading input graph..." << endl;
	auto readGraphstartTime = high_resolution_clock::now();//Time calculation starts
	read_graphEdges(AdjList, argv[1], &nodes);
	auto readGraphstopTime = high_resolution_clock::now();//Time calculation ends
	auto readGraphduration = duration_cast<microseconds>(readGraphstopTime - readGraphstartTime);// duration calculation
	cout << "Reading input graph completed" << endl;
	cout << "Time taken to read input graph: " << readGraphduration.count() << " microseconds" << endl;

	//Reading change edges input
	//vector<changeEdge> allChange;
	vector<changeEdge> allChange_Ins, allChange_Del;
	cout << "Reading input changed edges data..." << endl;
	auto readCEstartTime = high_resolution_clock::now();//Time calculation starts
	readin_changes(argv[5], allChange_Ins, allChange_Del, AdjList, totalInsertion);
	auto readCEstopTime = high_resolution_clock::now();//Time calculation ends
	auto readCEduration = duration_cast<microseconds>(readCEstopTime - readCEstartTime);// duration calculation
	cout << "Reading input changed edges data completed. totalInsertion:" << totalInsertion << endl;
	cout << "Time taken to read input changed edges: " << readCEduration.count() << " microseconds" << endl;

	//create 1D array from 2D to fit it in GPU
	cout << "creating 1D array from 2D to fit it in GPU" << endl;
	AdjListTracker[0] = 0; //start pointer points to the first index of InEdgesList
	for (int i = 0; i < nodes; i++) {
		AdjListTracker[i + 1] = AdjListTracker[i] + AdjList.at(i).size();
		AdjListFull.insert(std::end(AdjListFull), std::begin(AdjList.at(i)), std::end(AdjList.at(i)));
	}
	cout << "creating 1D array from 2D completed" << endl;


	//Transferring input graph and change edges data to GPU
	cout << "Transferring graph data from CPU to GPU" << endl;
	auto startTime_transfer = high_resolution_clock::now();
	ColWt* AdjListFull_device;
	cudaStatus = hipMallocManaged(&AdjListFull_device, (2 * (edges + totalInsertion)) * sizeof(ColWt));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at InEdgesListFull structure");
	}
	std::copy(AdjListFull.begin(), AdjListFull.end(), AdjListFull_device);

	int* AdjListTracker_device;
	cudaStatus = hipMalloc((void**)&AdjListTracker_device, (nodes + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at InEdgesListTracker_device");
	}
	hipMemcpy(AdjListTracker_device, AdjListTracker, (nodes + 1) * sizeof(int), hipMemcpyHostToDevice);

	int totalChangeEdges_Ins = allChange_Ins.size();
	changeEdge* allChange_Ins_device;
	cudaStatus = hipMallocManaged(&allChange_Ins_device, totalChangeEdges_Ins * sizeof(changeEdge));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at allChange_Ins structure");
	}
	std::copy(allChange_Ins.begin(), allChange_Ins.end(), allChange_Ins_device);

	int totalChangeEdges_Del = allChange_Del.size();
	changeEdge* allChange_Del_device;
	cudaStatus = hipMallocManaged(&allChange_Del_device, totalChangeEdges_Del * sizeof(changeEdge));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at allChange_Del structure");
	}
	std::copy(allChange_Del.begin(), allChange_Del.end(), allChange_Del_device);

	auto stopTime_transfer = high_resolution_clock::now();//Time calculation ends
	auto duration_transfer = duration_cast<microseconds>(stopTime_transfer - startTime_transfer);// duration calculation
	cout << "**Time taken to transfer graph data from CPU to GPU: "
		<< float(duration_transfer.count()) / 1000 << " milliseconds**" << endl;


	//set hipMemAdviseSetReadMostly by the GPU for change edge data
	hipMemAdvise(allChange_Ins_device, totalChangeEdges_Ins * sizeof(changeEdge), hipMemAdviseSetReadMostly, deviceId);
	hipMemAdvise(allChange_Del_device, totalChangeEdges_Del * sizeof(changeEdge), hipMemAdviseSetReadMostly, deviceId);


	//Reading SSSP Tree input and storing directly on unified memory
	RT_Vertex* SSSP;
	cudaStatus = hipMallocManaged(&SSSP, nodes * sizeof(RT_Vertex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSP structure");
	}
	cout << "Reading input SSSP tree data..." << endl;
	auto readSSSPstartTime = high_resolution_clock::now();//Time calculation starts
	read_SSSP(SSSP, argv[4], &nodes);
	auto readSSSPstopTime = high_resolution_clock::now();//Time calculation ends
	auto readSSSPduration = duration_cast<microseconds>(readSSSPstopTime - readSSSPstartTime);// duration calculation
	cout << "Reading input SSSP tree data completed" << endl;
	cout << "Time taken to read input input SSSP tree: " << readSSSPduration.count() << " microseconds" << endl;
	//set hipMemAdviseSetPreferredLocation at GPU for SSSP data
	hipMemAdvise(SSSP, nodes * sizeof(RT_Vertex), hipMemAdviseSetPreferredLocation, deviceId);

	//int inf = 999999;
	/*int* Edgedone;
	hipMallocManaged(&Edgedone, (totalChangeEdges) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSP structure");
	}
	//set hipMemAdviseSetPreferredLocation at GPU for Edge Done data
	hipMemAdvise(Edgedone, (totalChangeEdges) * sizeof(int), hipMemAdviseSetPreferredLocation, deviceId);
	*/

	//Asynchronous prefetching of data
	//hipMemPrefetchAsync(Edgedone, (totalChangeEdges) * sizeof(int), deviceId);
	hipMemPrefetchAsync(allChange_Ins_device, totalChangeEdges_Ins * sizeof(changeEdge), deviceId);
	hipMemPrefetchAsync(allChange_Del_device, totalChangeEdges_Del * sizeof(changeEdge), deviceId);
	hipMemPrefetchAsync(SSSP, nodes * sizeof(RT_Vertex), deviceId);
	hipMemPrefetchAsync(AdjListFull_device, edges * sizeof(ColWt), deviceId);
	//hipMemPrefetchAsync(OutEdgesListFull_device, edges * sizeof(ColWt), deviceId);
	//int* change_d = new int[1];
	int* change = 0;
	hipMallocManaged(&change, sizeof(int));
	//hipMalloc((void**)&change_d, 1 * sizeof(int));
	int its = 0;
	cout << "reading input data completed" << endl;
	int* affectedNodeList;
	hipMallocManaged(&affectedNodeList, nodes * sizeof(int));
	int* counter = 0;
	hipMallocManaged(&counter, sizeof(int));
	auto startTime1 = high_resolution_clock::now(); //Time calculation start
	//initialize Edgedone array with -1
	//initializeEdgedone << <(totalChangeEdges / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (Edgedone, totalChangeEdges);
	//process changed edges

	//processChangedEdges << <(totalChangeEdges / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (allChange_device, Edgedone, SSSP, totalChangeEdges, inf, AdjListFull_device, AdjListTracker_device, affectedNodeList, counter);
	//    hipDeviceSynchronize();



	//process change edges
	deleteEdge << < (totalChangeEdges_Del / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (allChange_Del_device, SSSP, totalChangeEdges_Del, AdjListFull_device, AdjListTracker_device);
	insertEdge << < (totalChangeEdges_Ins / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (allChange_Ins_device, SSSP, totalChangeEdges_Ins, AdjListFull_device, AdjListTracker_device);





	int* counter_del = 0;
	hipMallocManaged(&counter_del, sizeof(int));
	int* affectedNodeList_del;
	hipMallocManaged(&affectedNodeList_del, nodes * sizeof(int));
	int* updatedAffectedNodeList_del;
	hipMallocManaged(&updatedAffectedNodeList_del, nodes * sizeof(int));
	int* updated_counter_del = 0;
	hipMallocManaged(&updated_counter_del, sizeof(int));
	filterAffectedNodes << <(nodes / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (SSSP, affectedNodeList, counter, nodes, affectedNodeList_del, counter_del);
	hipDeviceSynchronize();

	*change = 1;

	while (*change == 1) {
		*change = 0;

		cout << "Only for Deletion=";
		for (int i = 0; i < *counter_del; i++)
		{
			cout << affectedNodeList_del[i];
		}
		cout << endl;

		updateNeighbors_del << <(*counter_del / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> >
			(SSSP, updated_counter_del, updatedAffectedNodeList_del, affectedNodeList_del, counter_del,
				affectedNodeList, counter, AdjListFull_device, AdjListTracker_device, change);
		hipDeviceSynchronize();


		*counter_del = *updated_counter_del;



		copyArray << <(*counter_del / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (updatedAffectedNodeList_del, counter_del, affectedNodeList_del);
		hipDeviceSynchronize();
		//        copy(begin(*updatedAffectedNodeList_del), end(*updatedAffectedNodeList_del), begin(*affectedNodeList_del));
		*updated_counter_del = 0;

	}


	cout << "affected node list: ";
	for (int i = 0; i < *counter; i++)
	{
		cout << affectedNodeList[i] << endl;
	}

	/*for (int i = 0; i < *counter_del; i++)
	{
		cout << "Only for Deletion=" << affectedNodeList_del[i] << endl;
	}*/



	//    auto stopTime1 = high_resolution_clock::now();//Time calculation ends
	//    auto duration1 = duration_cast<microseconds>(stopTime1 - startTime1);// duration calculation
	//    cout << "**Time taken for STEP 1: "
	//         << float(duration1.count()) / 1000 << " milliseconds**" << endl;
	//
	//    //Step 2 starts
	//    auto startTime2 = high_resolution_clock::now(); //Time calculation start
	//    change[0] = 1;
	//    while (change[0] == 1) {
	//        change[0] = 0;
	//        hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);
	//        updateNeighbors_del << <(nodes / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (SSSP, nodes, inf, AdjListFull_device, AdjListTracker_device, change_d);
	//        hipMemcpy(change, change_d, 1 * sizeof(int), hipMemcpyDeviceToHost);
	//        hipDeviceSynchronize();
	//    }
	//
	//    //update the distance of neighbors and connect the disconnected subgraphs
	//    change[0] = 1;
	//    while (change[0] == 1) {
	//        change[0] = 0;
	//        hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);
	//        updateNeighbors << <(nodes / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (SSSP, nodes, inf, AdjListFull_device, AdjListTracker_device, change_d);
	//        hipMemcpy(change, change_d, 1 * sizeof(int), hipMemcpyDeviceToHost);
	//        hipDeviceSynchronize();
	//        its++;
	//    }
	//    auto stopTime2 = high_resolution_clock::now();//Time calculation ends
	//    auto duration2 = duration_cast<microseconds>(stopTime2 - startTime2);// duration calculation
	//    cout << "**Time taken for STEP 2: "
	//         << float(duration2.count()) / 1000 << " milliseconds**" << endl;
	//    printf("Total Iterations to Converge %d \n", its);
	//    cout << "****Total Time taken for SSSP update: "
	//         << float(duration1.count() + duration2.count()) / 1000 << " milliseconds****" << endl;
	//
	//
	//    //print output:
	//    printSSSP << <1, 1 >> > (SSSP, nodes);
	//    hipDeviceSynchronize();
	//    int x;
	//    if (nodes < 40) {
	//        x = nodes;
	//    }
	//    else {
	//        x = 40;
	//    }
	//    cout << "from CPU: \n[";
	//    for (int i = 0; i < x; i++) {
	//        cout << i << ":" << SSSP[i].Dist << " ";
	//    }
	//    cout << "]\n";
		//print output ends



	//hipFree(change_d);
	hipFree(change);
	hipFree(affectedNodeList);
	hipFree(affectedNodeList_del);
	hipFree(updatedAffectedNodeList_del);
	hipFree(counter);
	hipFree(counter_del);
	hipFree(updated_counter_del);
	hipFree(AdjListTracker_device);
	hipFree(AdjListTracker_device);
	hipFree(allChange_Del_device);
	hipFree(allChange_Ins_device);
	hipFree(SSSP);
	return 0;
}